#include "hip/hip_runtime.h"
/****This program uses map of CUDA
 to convert color image to grayscale image*****/

#include <hip/hip_runtime.h>
#include <device_laumch_parameters.h>
#include <stdio.h>

__global__ void color2gray(int width,int height,int *picr_d,int *picg_d,int *picb_d,int *picGray_d)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<width*height)
		picGray_d[i]=(int)(picr_d[i] * 0.299 + picg_d[i] * 0.587 + 0.114 * picb_d[i]);
}

int main(int argc, char *argv[])
{
	int width;
	int height;
	FILE *f;
	int *picr_h; //red channel host
	int *picb_h; //blue channel host
	int *picg_h; //green channel host
	
	int *picr_d; //red channel device
	int *picb_d; //blue channel device
	int *picg_d; //green channel device
	int *picGray_d; //grayscale image device
	int i,j;
	char *str;
	
	
	width=atoi(argv[1]);
	height=atoi(argv[2]);
	
	picr_h=(int*)malloc(sizeof(int)*width*height);	
	
	picb_h=(int*)malloc(sizeof(int)*width*height);	
		
	picg_h=(int*)malloc(sizeof(int)*width*height);
	
	str=(char*)malloc(sizeof(char)*width);
	
	hipMalloc(&picGray_d,sizeof(int)*width*height);
	hipMalloc(&picr_d,sizeof(int)*width*height);
	hipMalloc(&picb_d,sizeof(int)*width*height);
	hipMalloc(&picg_d,sizeof(int)*width*height);
	
	//reading a ppm file
	f=fopen(argv[3],"r");
	fgets(str,width,f);
	fgets(str,width,f);
	fgets(str,width,f);
	for(i=0;i<height;i++){
		fgets(str,width,f);
		for(j=0;j<width;j++){
			sscanf(str,"%d",&picr_h[j+i*height]);
			sscanf(str,"%d",&picg_h[j+i*height]);
			sscanf(str,"%d",&picb_h[j+i*height]);
		}
	}
	fclose(f);
		
	
	hipMemcpy(picr_d, picr_h, width*height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(picb_d, picb_h, width*height*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(picg_d, picg_h, width*height*sizeof(int), hipMemcpyHostToDevice);
	
	
	
	//color2gray kernel launch
	color2gray<<<height,width>>>(width,height,picr_d,picg_d,picb_d,picGray_d);
	
	hipMemcpy(picr_h, picGray_d, width*height*sizeof(int), hipMemcpyDeviceToHost);
	
	f=fopen(argv[4],"w");
	fprintf(f,"P3\n%d %d\n255\n",width,height);
	for(i=0;i<height;i++){
		for(j=0;j<width;j++){
			fprintf(f,"%d %d %d ",picr_h[j+i*height],picr_h[j+i*height],picr_h[j+i*height]);
		}
		fprintf(f,"\n");
	}
	
	fclose(f);
	
	hipFree(picr_d);
	hipFree(picg_d);
	hipFree(picb_d);
	hipFree(picGray_d);
	
	free(picr_h);
	free(picg_h);
	free(picb_h);
	
}
	
	
	
	
